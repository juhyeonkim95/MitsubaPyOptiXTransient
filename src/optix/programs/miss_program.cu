#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "optix/common/prd_struct.h"
#include "optix/light/light_parameters.h"
#include "optix/common/helpers.h"

using namespace optix;

rtDeclareVariable(SurfaceInteraction, si, rtPayload, );
rtDeclareVariable(float3, bg_color, , );
rtBuffer<LightParameter> sysLightParameters;
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_PROGRAM void miss()
{
    si.is_valid = false;
}

RT_PROGRAM void miss_environment_mapping()
{
    si.is_valid = false;
    LightParameter& light = sysLightParameters[0];

    float3 ray_direction = transform_normal(light.transformation.transpose(), ray.direction);

    float phi = atan2f(ray_direction.x, -ray_direction.z);
    float theta = acosf(-ray_direction.y);
    float u = (phi + M_PIf) * (0.5f * M_1_PIf);
    float v = theta * M_1_PIf;

    si.emission = make_float3(optix::rtTex2D<float4>(light.envmapID , u, v));
}