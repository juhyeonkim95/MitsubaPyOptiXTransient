#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optix/common/prd_struct.h"
#include "optix/common/helpers.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );

rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );


rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(SurfaceInteraction, si, rtPayload, );

rtBuffer<MaterialParameter> sysMaterialParameters;
rtDeclareVariable(int, materialId, , );

RT_PROGRAM void closest_hit()
{
    // Transform normal from object to world coordinate
	const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal ) );

    // Face forwarding normal (ffnormal dot ray_direction > 0)
    float3 ff_normal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    // Material parameter
	MaterialParameter& mat = sysMaterialParameters[materialId];
    float3 normal = mat.isTwosided? ff_normal : world_shading_normal;

    si.material_id = materialId;
    si.normal = normal;
    si.uv = texcoord;

    float3 wi = -ray.direction;
    optix::Onb onb( normal );

    si.wi = to_local(onb, wi);
    si.p = ray.origin + t_hit * ray.direction;
    si.t = t_hit;
}